//this program is modified from http://coyagi.tistory.com/entry/c-OpenCV-%EC%82%AC%EC%A7%84%ED%9D%91%EB%B0%B1%EB%B3%80%ED%99%981
//Created by hojae Son 10/20/2016

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<opencv2/opencv.hpp>  
#include "" 
#include <iostream>

using namespace cv;
using namespace std;

void toGrayScale(IplImage* frame);

void main()
{ 
	CvCapture* capture = cvCreateFileCapture("C:\\Users\\pancake\\Desktop\\test.avi"); // VIDEO PATH FROM LOCAL

	IplImage* frame;
	char c; 
	while (1){
		frame = cvQueryFrame(capture);
		if (!frame) break;  
		toGrayScale(frame); //ACTUAL FUNCTION
		c = cvWaitKey(33);
		if (c == 27) break;
	} 
	  
	cvWaitKey(0);
	cvReleaseImage(&frame); 
}

void toGrayScale(IplImage* frame){ 
	//original video
	cvNamedWindow("Original", CV_WINDOW_AUTOSIZE);
	cvShowImage("Original", frame);

	//grayScale video processing
	int  intensity; 
	uchar* data, *grayScale_data; 
	data = (uchar*)frame->imageData;

	frame = frame;  
	grayScale_data = (uchar*)frame->imageData;

	for (int i = 0; i < frame->height; i++)
		for (int j = 0; j < frame->width; j++)
		{
			intensity = (int)(0.114 * data[i* frame->width*frame->nChannels + j*frame->nChannels + 0]
				+ 0.587 * data[i* frame->width*frame->nChannels + j*frame->nChannels + 1]
				+ 0.299 * data[i* frame->width*frame->nChannels + j*frame->nChannels + 2]);
			for (int k = 0; k < frame->nChannels; k++)
				grayScale_data[i* frame->width*frame->nChannels + j*frame->nChannels + k] = intensity;
		}

	for (int i = 0; i < 1000; i += 100)
		for (int k = 0; k < 3; k++)
			printf("imageData[%3d] = %d \t frameData[%3d] = %d\n", i + k, data[i + k], i + k, grayScale_data[i + k]);

	cvNamedWindow("GrayScale", CV_WINDOW_AUTOSIZE);
	cvShowImage("GrayScale", frame); 
}

 